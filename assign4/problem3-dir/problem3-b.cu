#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;

#define NSEC_SEC_MUL (1.0e9)
#define ITER_CHUNK_SIZE (1 << 25)
#define NUM_VAR 10
#define THRESHOLD (std::numeric_limits<double>::epsilon())

#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Kernel to compute valid results and store x arrays in ordered buffer
__global__ void kernel(double *constraints, long long *loop_iter, double *dev_a, double *dev_b, long long *dev_output_x, int *dev_output_count, long long chunk_start, long long chunk_end) {
    int tx = threadIdx.x;
    int x = blockIdx.x * blockDim.x + tx;  // Global thread ID
    long long total_threads = gridDim.x * blockDim.x;
    long long iter_per_thread = (chunk_end - chunk_start + total_threads - 1) / total_threads;
    long long start_iter = chunk_start + iter_per_thread * x;
    long long end_iter = min(chunk_end - 1, start_iter + iter_per_thread - 1);

    double x_array[NUM_VAR];
    double q[NUM_VAR] = {0.0};
    long long iter_no[NUM_VAR];

    for (long long iter = start_iter; iter <= end_iter; iter++) {
        long long tmp_iter = iter;

        for (int i = NUM_VAR - 1; i >= 0; i--) {
            iter_no[i] = tmp_iter % loop_iter[i];
            tmp_iter /= loop_iter[i];
            x_array[i] = dev_b[3 * i] + iter_no[i] * dev_b[3 * i + 2];
        }

        bool is_valid = true;
        for (int i = 0; i < NUM_VAR; i++) {
            q[i] = 0.0;
            // #pragma unroll
            for (int j = 0; j < NUM_VAR; j++) {
                q[i] += dev_a[i * 12 + j] * x_array[j];
            }
            q[i] -= dev_a[i * 12 + 10];
            is_valid &= (fabs(q[i]) <= constraints[i]);
        }
        if (is_valid) {
          int old = atomicAdd(dev_output_count, 1); 
          dev_output_x[old] = iter;
        }
    }
}

int main() {
    double *a, *b;
    int i, j;
    cudaCheckError(hipMallocManaged(&a, 120*sizeof(double)));
    cudaCheckError(hipMallocManaged(&b, 30*sizeof(double)));

    FILE* fp = fopen("./disp.txt", "r");
    if (fp == NULL) {
        printf("Error: could not open file\n");
        return 1;
    }
    for (i = 0; !feof(fp) && i < 120; i++) {
        if (!fscanf(fp, "%lf", &a[i])) {
            printf("Error reading disp.txt\n");
            exit(EXIT_FAILURE);
        }
    }
    fclose(fp);

    FILE* fpq = fopen("./grid.txt", "r");
    if (fpq == NULL) {
        printf("Error: could not open file\n");
        return 1;
    }
    for (j = 0; !feof(fpq) && j < 30; j++) {
        if (!fscanf(fpq, "%lf", &b[j])) {
            printf("Error reading grid.txt\n");
            exit(EXIT_FAILURE);
        }
    }
    fclose(fpq);

    double kk = 0.3;
    double *constraints;
    cudaCheckError(hipMallocManaged(&constraints, NUM_VAR * sizeof(double)));
    for (i = 0; i < NUM_VAR; i++) {
        constraints[i] = kk * a[11 + i * 12];
    }

    long long *loop_iter;
    cudaCheckError(hipMallocManaged(&loop_iter, (NUM_VAR+1)*sizeof(double)));
    long long total_iter = 1;
    for (i = 0; i < NUM_VAR; i++) {
        loop_iter[i] = floor((b[3 * i + 1] - b[3 * i]) / b[3 * i + 2]);
        total_iter *= loop_iter[i];
    }
    loop_iter[NUM_VAR] = total_iter;
    int device;
    hipGetDevice(&device);
    hipMemPrefetchAsync(a, 120 * sizeof(double), device);
    hipMemPrefetchAsync(b, 30 * sizeof(double), device);
    cudaCheckError(hipMemAdvise(a, 120 * sizeof(double), hipMemAdviseSetReadMostly, device));
    cudaCheckError(hipMemAdvise(b, 30 * sizeof(double), hipMemAdviseSetReadMostly, device));
    cudaCheckError(hipMemAdvise(constraints, NUM_VAR * sizeof(double), hipMemAdviseSetReadMostly, device));
    cudaCheckError(hipMemAdvise(loop_iter, (NUM_VAR + 1) * sizeof(long long), hipMemAdviseSetReadMostly, device));

    long long result_cnt = 0;
    int *output_count;
    long long *output_x;
    hipMallocManaged(&output_count, sizeof(int));
    hipMallocManaged(&output_x, ITER_CHUNK_SIZE * sizeof(long long));

    int block = 512;
    int grid = (1 << 16);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    ofstream output_file("results-vb.txt");
    output_file << setprecision(6) << fixed;

    for (long long chunk_start = 0; chunk_start < total_iter; chunk_start += ITER_CHUNK_SIZE) {
        long long chunk_end = min(chunk_start + ITER_CHUNK_SIZE, total_iter);

        *output_count = 0;
        kernel<<<grid, block>>>(constraints, loop_iter, a, b, output_x, output_count, chunk_start, chunk_end);
        hipDeviceSynchronize();
        
        result_cnt += (*output_count);
        if (*output_count > 0) {
            sort(output_x, output_x + (*output_count));
            for (int k = 0; k < (*output_count); k ++) {
                double x_array[NUM_VAR];
                long long tmp_iter = output_x[k];
                for(int i=NUM_VAR-1; i>=0; i--){
                    x_array[i] = b[3 * i] + (tmp_iter % loop_iter[i]) * b[3 * i + 2];
                    tmp_iter /= loop_iter[i];
                }
                for (int l = 0; l < NUM_VAR; l++) {
                    output_file << x_array[l];
                    if(l == NUM_VAR-1)
                      output_file << std::endl;
                    else 
                      output_file << "\t";
                }
            }
        }
    }

    output_file.close();

    hipEventRecord(end);
    hipDeviceSynchronize();
    float kernel_time = 0.0;
    hipEventElapsedTime(&kernel_time, start, end);
    std::cout << "Kernel time " << kernel_time * 1e-3 << "s\n";
    std::cout << "Result pnts " << result_cnt << std::endl;
  
    hipFree(a);
    hipFree(b);
    hipFree(constraints);
    hipFree(loop_iter);
    hipFree(output_count);
    hipFree(output_x);
    return EXIT_SUCCESS;
}






//   int *pts_devs[num_streams];
//   int *device_bool_arrs[num_streams];
//   hipEvent_t starts[num_streams];
//   hipEvent_t stops[num_streams];

//   for(int i = 0; i < num_streams; i++){
//     cudaCheckError(hipStreamCreate(&streams[i]));
//   }

//   for(long i = 0; i<iter; i+= ITER_SIZE * num_streams){

//     for(int j = 0; j < num_streams; j++){
//       tmp_olds[j] = tmps[j];
//       if(i == 0) cudaCheckError(hipMalloc(&pts_devs[j], 1 * sizeof(int)));
//       cudaCheckError(hipMemcpyAsync(pts_devs[j], &tmps[j], 1 * sizeof(int), hipMemcpyHostToDevice, streams[j]));
//       if(i == 0) cudaCheckError(hipMalloc(&device_bool_arrs[j], ITER_SIZE * sizeof(int)));
//       cudaCheckError(hipMemcpyAsync(device_bool_arrs[j], bool_arr_zero, ITER_SIZE * sizeof(int), hipMemcpyHostToDevice, streams[j]));

//       hipEventCreate(&starts[j]);
//       hipEventCreate(&stops[j]);
    
//       hipEventRecord(starts[j], streams[j]);

//       trial<<<4096, 64, 0, streams[j]>>>(i + j * ITER_SIZE, ((long)ITER_SIZE)/((long)(4096*64)), iter, s_dev, dd_dev, c_dev, d_dev, e_dev, pts_devs[j], device_bool_arrs[j]);

//       hipEventRecord(stops[j], streams[j]);
//     }

//     // int tmp_old = tmp;

//     // hipEventRecord(start);
//     // trial<<<4096, 64>>>(i, ((long)ITER_SIZE)/((long)(4096*64)), iter, s_dev, dd_dev, c_dev, d_dev, e_dev, pts_dev, device_bool_arr);

//     // hipEventRecord(stop);
//     // hipEventSynchronize(stop);
//     // cudaCheckError(hipDeviceSynchronize());

//     for(int j = 0; j < num_streams; j++){
//       hipEventSynchronize(stops[j]);
//       float milliseconds = 0;
//       hipEventElapsedTime(&milliseconds, starts[j], stops[j]);
//       total_time += milliseconds;

//       cudaCheckError(hipMemcpyAsync(&tmps[j], pts_devs[j], 1 * sizeof(int), hipMemcpyDeviceToHost, streams[j]));
//       cudaCheckError(hipStreamSynchronize(streams[j]));

//       if(tmps[j] != tmp_olds[j]){
//         cudaCheckError(hipMemcpyAsync(bool_arr, device_bool_arrs[j], ITER_SIZE * sizeof(int), hipMemcpyDeviceToHost, streams[j]));
//         cudaCheckError(hipStreamSynchronize(streams[j]));
//         cudaCheckError(hipMemcpyAsync(device_bool_arrs[j], bool_arr_zero, ITER_SIZE * sizeof(int), hipMemcpyHostToDevice, streams[j]));

//         process_points(bool_arr, i + j * ITER_SIZE, fptr, dd, s);

//         cudaCheckError(hipStreamSynchronize(streams[j]));
//       }
//     }


//     // cudaCheckError(hipMemcpy(&tmp, pts_dev, 1 * sizeof(int), hipMemcpyDeviceToHost));
//     // if(tmp != tmp_old){
//     //   cudaCheckError(hipMemcpy(bool_arr, device_bool_arr, ITER_SIZE * sizeof(int), hipMemcpyDeviceToHost));
//     //   cudaCheckError(hipMemcpy(device_bool_arr, bool_arr_zero, ITER_SIZE * sizeof(int), hipMemcpyHostToDevice));

//     //   process_points(bool_arr, i, fptr, dd, s);
//     // }
//   }
