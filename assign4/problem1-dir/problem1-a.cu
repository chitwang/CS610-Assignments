#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <sys/time.h>
#include <cmath>

#define THRESHOLD (std::numeric_limits<double>::epsilon())

using std::cerr;
using std::cout;
using std::endl;

#define cudaCheckError(ans)                                                    \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

const uint64_t N = (64);
const uint64_t TILE = 4;

// TODO: Edit the function definition as required
__global__ void kernel1(const double *in, double *out) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;

  if (i > 0 && i < N - 1 && j > 0 && j < N - 1 && k > 0 && k < N - 1) {
    out[i * N * N + j * N + k] = 0.8 * (in[(i - 1) * N * N + j * N + k] +
                                        in[(i + 1) * N * N + j * N + k] +
                                        in[i * N * N + (j - 1) * N + k] +
                                        in[i * N * N + (j + 1) * N + k] +
                                        in[i * N * N + j * N + (k - 1)] +
                                        in[i * N * N + j * N + (k + 1)]);
  }
}

// TODO: Edit the function definition as required
__host__ void stencil(const double *in, double *out) {
  for (uint64_t i =1; i<N -1; i++) {
    for (uint64_t j =1; j<N -1; j++) {
      for (uint64_t k =1; k<N -1; k++) {
        out[i * N * N + j * N + k] = 0.8 * (in[(i - 1) * N * N + j * N + k] +
                                            in[(i + 1) * N * N + j * N + k] +
                                            in[i * N * N + (j - 1) * N + k] +
                                            in[i * N * N + (j + 1) * N + k] +
                                            in[i * N * N + j * N + (k - 1)] +
                                            in[i * N * N + j * N + (k + 1)]);
      }
    }
  }
}

__host__ void check_result(const double* w_ref, const double* w_opt,
                           const uint64_t size) {
  double maxdiff = 0.0;
  int numdiffs = 0;

  for (uint64_t i = 0; i < size; i++) {
    for (uint64_t j = 0; j < size; j++) {
      for (uint64_t k = 0; k < size; k++) {
        double this_diff =
            w_ref[i + N * j + N * N * k] - w_opt[i + N * j + N * N * k];
        if (std::fabs(this_diff) > THRESHOLD) {
          numdiffs++;
          
          if (fabs(this_diff) > maxdiff) {
            maxdiff = fabs(this_diff);
          }
        }
      }
    }
  }

  if (numdiffs > 0) {
    cout << numdiffs << " Diffs found over THRESHOLD " << THRESHOLD
         << "; Max Diff = " << maxdiff << endl;
  } else {
    cout << "No differences found between base and test versions\n";
  }
}

void print_mat(const double* A) {
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      for (int k = 0; k < N; ++k) {
        printf("%lf,", A[i * N * N + j * N + k]);
      }
      printf("      ");
    }
    printf("\n");
  }
}

double rtclock() { // Seconds
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday(&Tp, &Tzp);
  if (stat != 0) {
    cout << "Error return from gettimeofday: " << stat << "\n";
  }
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

int main() {
  uint64_t SIZE = N * N * N;
  std::cout << "The program will run CPU version and a naive CUDA Kernel for stencil\n#############Results##############\n";
  // Initialisation
  auto *h_in = new double[SIZE];
  auto *h_out = new double[SIZE];
  auto *cuda_out = new double[SIZE];
  
  for(uint64_t i=0;i<SIZE;i++){
    h_in[i] = static_cast<double>(rand())/RAND_MAX;
  }

  double clkbegin = rtclock();
  stencil(h_in, h_out);
  double clkend = rtclock();
  double cpu_time = clkend - clkbegin;
  cout << "Stencil time on CPU: " << cpu_time * 1000 << " msec" << endl;

  hipError_t status;
  hipEvent_t start, end;
  hipEvent_t start_kernel, end_kernel;
  float kernel_time, overall_time;

  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventCreate(&start_kernel);
  hipEventCreate(&end_kernel);
  
  double *d_in, *d_out;
  cudaCheckError(hipMalloc(&d_in, SIZE * sizeof(double)));
  cudaCheckError(hipMalloc(&d_out, SIZE * sizeof(double)));

  hipEventRecord(start);
  // Copy input to GPU
  cudaCheckError(hipMemcpy(d_in, h_in, SIZE * sizeof(double), hipMemcpyHostToDevice));

  dim3 block(8, 8, 8);
  dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y, (N + block.z - 1) / block.z);

  hipEventRecord(start_kernel);
  kernel1<<<grid, block>>>(d_in, d_out);
  hipEventRecord(end_kernel);
  cudaCheckError(hipDeviceSynchronize());
  hipEventElapsedTime(&kernel_time, start_kernel, end_kernel);

  cudaCheckError(hipMemcpy(cuda_out, d_out, SIZE * sizeof(double), hipMemcpyDeviceToHost));
  hipEventRecord(end);
  
  cudaCheckError(hipDeviceSynchronize());
  hipEventElapsedTime(&overall_time, start, end);
  std::cout << "Only Kernel time: " << kernel_time << "ms\n";
  std::cout << "Overall time: " << overall_time << "ms\n";

  check_result(cuda_out, h_out, N);

  hipEventDestroy(start);
  hipEventDestroy(end);
  hipEventDestroy(start_kernel);
  hipEventDestroy(end_kernel);

  // Free memory
  hipFree(d_in);
  hipFree(d_out);

  // TODO: Free memory
  delete [] h_in;
  delete [] h_out;
  delete [] cuda_out;

  return EXIT_SUCCESS;
}
